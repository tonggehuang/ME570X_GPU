#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <ctime>
#include <string>
#include <fstream>
#include <math.h>

using namespace std;

#define rawR 7
#define rawC 840
#define rawL (rawR*rawC)
#define LENGTH 840
#define BATCH 1
#define LENGTHPAD 1024
#define NRANK 2

static __global__ void cufftComplexScale(hipfftComplex *idata, hipfftComplex *odata, const int size, float scale)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadID < size)
    {
        odata[threadID].x = idata[threadID].x * scale;
        odata[threadID].y = idata[threadID].y * scale;
    }
}

int main()
{
  int n[NRANK] = {rawR, rawC};
  // create arrays
  float speed2d[rawR][rawC];

  // read raw data 
  ifstream rawData("../data/speedData.txt");

  // generate 2d speed data
  if (!(rawData.is_open())){
    cout<<"faild to read data." << endl;
  } 

  for (int row=0; row<rawR; row++){
    for (int col=0; col<rawC; col++){
      rawData >> speed2d[row][col];
    }
  }

  rawData.close();

  // print array for debug
  // for (int row=0; row<rawR; row++){
  //   for (int col=0; col<10; col++){
  //     cout << speed2d[row][col] << '\t';
  //   }
  //   cout << '\n';
  // }

  // host data pointer
  hipfftComplex *CompData2d=(hipfftComplex*)malloc(rawC*rawR*sizeof(hipfftComplex)); 

  // 2 d

  for (int i=0; i<rawR; i++){
    for (int j=0; j<rawC; j++){
      CompData2d[i*rawC+j].x = speed2d[i][j];
      CompData2d[i*rawC+j].y = 0;
    }
  }

  hipfftComplex *d_fftData; // device data pointer
  hipMalloc((void**)&d_fftData,rawC*rawR*sizeof(hipfftComplex));
  hipMemcpy(d_fftData,CompData2d,rawC*rawR*sizeof(hipfftComplex),hipMemcpyHostToDevice);

  // create the cuda event to count the running time for GPU
  hipEvent_t start1, stop1;
  hipEventCreate(&start1);
  hipEventCreate(&stop1);

  hipfftHandle plan;
  hipfftPlanMany(&plan, NRANK, n,
        NULL, 1, 0,
        NULL, 1, 0,
        HIPFFT_C2C, BATCH);

  // execute kernel
  

  hipfftExecC2C(plan,(hipfftComplex*)d_fftData,(hipfftComplex*)d_fftData,HIPFFT_FORWARD);


  hipEventRecord(start1);
  hipfftExecC2C(plan, (hipfftComplex*)d_fftData, (hipfftComplex*)d_fftData, HIPFFT_BACKWARD);
 
  dim3 dimBlock(1024);
  dim3 dimGrid(6); 

  cufftComplexScale <<<dimGrid, dimBlock>>>((hipfftComplex*)d_fftData,(hipfftComplex*)d_fftData,rawC*rawR,1.0f / (rawC*rawR));
  
  hipEventRecord(stop1);

  hipEventSynchronize(stop1);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start1, stop1);
  cout << "GPU FFT time used: "<< milliseconds << " ms\n";



  hipDeviceSynchronize();

  hipMemcpy(CompData2d,d_fftData,rawC*rawR*sizeof(hipfftComplex)*BATCH,hipMemcpyDeviceToHost);

  //store result to txt
  ofstream myFile;
  myFile.open("../data/2d_batch_inverse.txt");

  for (int i=0; i<rawR; i++){
    for (int j=0; j<rawC; j++){
      myFile << CompData2d[i*rawC+j].x <<','<< CompData2d[i*rawC+j].y << endl;
    }
  }

  hipfftDestroy(plan);
  free(CompData2d);
  hipFree(d_fftData);

  return 0;
}